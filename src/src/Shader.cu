#include "hip/hip_runtime.h"
#include "../Graphics/opengl/Shader.h"

// Create vertex/ fragment shaders from GLSL files
Shader::Shader(const char* inputVertex, const char* inputFragment, Log& log) {
    log.new_state("Shader");
    unsigned int vertex, fragment;
    success = true;
    log.log_state("Parsing:" + std::string(inputVertex));
    log.log_state("Parsing:" + std::string(inputFragment));
    char* source = Shader::Parse(inputVertex, log);
    char* source2 = Shader::Parse(inputFragment, log);
    std::string type1 = "vertex shader";
    std::string type2 = "fragment shader";
    BuildShaders(vertex, source, GL_VERTEX_SHADER, type1, log);
    BuildShaders(fragment, source2, GL_FRAGMENT_SHADER, type2, log);
    ID = glCreateProgram();
    glAttachShader(ID, vertex);
    glAttachShader(ID, fragment);
    Link(ID, log);
    free(source);
    free(source2);
}
Shader::Shader(const char* inputCompute, Log& log) {
    log.new_state("Shader");
    unsigned int compute;
    std::string type = "compute";
    success = true;
    log.log_state("Parsing:" + std::string(inputCompute));
    char* source = Shader::Parse(inputCompute, log);
    BuildShaders(compute, source, GL_COMPUTE_SHADER, type, log);
    ID = glCreateProgram();
    glAttachShader(ID, compute);
    Link(ID, log);
    free(source);
}

Shader::~Shader() {
    glDeleteProgram(ID);
}
char* Shader::Parse(string input, Log& log) {
    ifstream in(input);
    try{
        !in.is_open() ? throw std::runtime_error("ERR: GLSL FILE NOT FOUND") : NULL;
    }
    catch (const std::runtime_error& e){
        log.log_state("Runtime Error: "  + std::string(e.what()));
        log.log_file.close();
        //rethrow
        throw std::runtime_error("ERR: GLSL FILE NOT FOUND");
    }
    std::stringstream buffer;
    buffer << in.rdbuf();
    string s = buffer.str();
    unsigned int alloc_size = sizeof(char) * (s.length() + 1);
    char* out = (char*)malloc(alloc_size);
    log.log_mem(alloc_size);
    for (int i = 0; i < s.length(); i++) {
        out[i] = s[i];
    }
    out[s.length()] = '\0';
    return out;
}
void Shader::BuildShaders(unsigned int& shader, const char* source, uint32_t shader_type, std::string& type, Log& log) {
    shader = glCreateShader(shader_type);
    glShaderSource(shader, 1, &source, NULL);
    int compile_status_local;
    char err[512];
    glCompileShader(shader);
    glGetShaderiv(shader, GL_COMPILE_STATUS, &compile_status_local);
    try{
        if (!compile_status_local) {
            glGetShaderInfoLog(shader, 512, NULL, err);
            log.log_state("Compilation failed: " + type);
            log.log_state(err);
            throw std::runtime_error("ERR: GLSL FILE NOT FOUND");
        }
    }
    catch (const std::runtime_error& e){
        this->success = false;
        log.log_state("Runtime Error"  + std::string(e.what()));
        log.log_file.close();
        //rethrow
        throw std::runtime_error("ERR: GLSL FILE NOT FOUND");
    }
}

void Shader::Link(unsigned int& program, Log& log) {
    glLinkProgram(program);
    int compile_status_local;
    char err[512];
    glGetProgramiv(program, GL_LINK_STATUS, &compile_status_local);
    try{
        if (!compile_status_local) {
            glGetProgramInfoLog(program, 512, NULL, err);
            log.log_state("Linking failed");
            log.log_state(err);
            throw std::runtime_error("ERR: GLSL FILE NOT FOUND");
        }
    }
    catch(const std::runtime_error& e){
        this->success = false;
        log.log_state("Runtime Error : " + std::string(e.what()));
        log.log_file.close();
        //rethrow
        throw std::runtime_error("ERR: GLSL FILE NOT FOUND");
    }
}


