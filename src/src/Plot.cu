//
// Created by domin on 09/05/2025.
//

#include "../Graphics/imgui/Plot.h"



Plot::Plot(const char* name, int x, int y) {
    this->plotX = (float*)malloc(sizeof(float) * x);
    this->plotY = (float*)malloc(sizeof(float) * y);
    this->mem_init = true;
    for (int i = 0; i < x; i++) {
        this->plotX[i] = i;
        this->plotY[i] = 0;
    }
    this->name = name;
}

Plot::Plot() {
    this->active = false;
    this->mem_init = false;
    this->x = 800;
    this->y = 800;
}

Plot::Plot(const Plot &rhs) {
    this->plotX = (float*)malloc(sizeof(float) * x);
    this->plotY = (float*)malloc(sizeof(float) * y);
    this->mem_init = true;
    this->name =rhs.name;
    memcpy(plotX, rhs.plotX, sizeof(float) * x);
    memcpy(plotY, rhs.plotY, sizeof(float) * y);
}

Plot& Plot::operator=(const Plot& other){
    if(this == & other) return * this;
    this->x = other.x;
    this->y = other.y;
    this->plotX = (float*)malloc(sizeof(float) * x);
    this->plotY = (float*)malloc(sizeof(float) * y);
    this->mem_init = true;
    this->autofit = other.autofit;
    this->active = other.active;
    memcpy(this->plotX, other.plotX, sizeof(float ) * other.x);
    memcpy(this->plotY, other.plotY, sizeof(float ) * other.y);
    return *this;
}

Plot::~Plot() {
    if(this->mem_init){
        free(plotX);
        free(plotY);
        this->mem_init = false;
    }
}

void Plot::RenderPlot(int n, const char* name) {
    ImPlot::SetNextAxesToFit();
    if (ImPlot::BeginPlot(name)) {
        if(!active){
            ImPlotRect limits = ImPlot::GetPlotLimits();
            min_lim = limits.X;
            max_lim = limits.Y;
            active = !active;
        }
        ImPlot::PlotLine(name, plotX, plotY, n);
        ImPlot::EndPlot();

    }
}
/*___________________________________MENU______________________________________*/

Menu::Menu(GLFWwindow* window, int x, int y) {
    setup_plotting(window);
    this->n = 400;
    this->autofit = false;
    plots = (Plot*)malloc(sizeof(Plot) * 10);
    active = (bool*)calloc(10, sizeof(bool));
    this->mem_init = true;
}
Menu::~Menu() {
    if(this->mem_init){
        free(plots);
        free(active);
        this->mem_init = false;
    }
}


void Menu::setup_plotting(GLFWwindow* window) {
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGui::StyleColorsDark();

    // Setup Platform/Renderer bindings
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330");
    ImPlot::CreateContext();
}

void Menu::ShowSettingsWindow() {
    ImGui::SetNextWindowPos(ImVec2(0, 0));
    ImGui::Begin("SETTINGS");

    // Display settings
    ImGui::Text("DISPLAY");
    ImGui::Checkbox("relaxation", &rel);
    if (rel) {
        ImGui::SliderFloat("Light Intensity", &tau, 0.0f, 2.0f);
    }
    ImGui::Separator();
    ImGui::Checkbox("Density", &rho);
    ImGui::Checkbox("Pressure", &p);
    ImGui::Separator();
    ImGui::Checkbox("Simulate", &simulation_state);
    ImGui::End();
}



void Menu::render_menu(GLFWwindow* window, Domain* d) {
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
    glfwMakeContextCurrent(window);
    ImGui_ImplOpenGL3_NewFrame();
    ImGui_ImplGlfw_NewFrame();
    ImGui::NewFrame();

    // Create a window for plotting
    ShowSettingsWindow();

    if (rho) {
        //transfer data from the solver to the plotter
        //hipMemcpy(this->plots[0].plotY, this->d->rho + 200 * 400, sizeof(float) * this->n, hipMemcpyDeviceToHost);
        ImGui::Begin("Density");
        if (!active[0]) {
            active[0] = !active[0];
            plots[0] = Plot("Density");
        }
        upload_data((void*)(plots[0].plotY), (void*)(d->rho + 400), 400 * sizeof(float));
        plots[0].RenderPlot(400, "Density");
        ImGui::End();
    }else{
        if (active[0]){
            //plots[0].~Plot();
            active[0] = !active[0];
        }
    }
    // Rendering
    ImGui::Render();
    glClearColor(0.0f, 0.0f, 0.0f, 1.00f);
    glClear(GL_COLOR_BUFFER_BIT);

    ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
    glfwSwapBuffers(window);
    glfwPollEvents();
}

void Menu::upload_data(void *dest, void *src, unsigned int size) {
    hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
}