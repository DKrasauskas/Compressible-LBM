#include "hip/hip_runtime.h"
//
// Created by domin on 11/05/2025.
//

#include "../Physics/dataset.cuh"
DatasetFP32::DatasetFP32(long long X, long long Y, int d, int q) {
    xy = X * Y;
    float** f_host = (float**)malloc( sizeof(float*) * q);
    float** f_host_buffer = (float**)malloc( sizeof(float*) * q);
    for(int i = 0 ; i < q; i ++){
        hipMalloc(&f_host[i], sizeof(float) * xy);
        hipMalloc(&f_host_buffer[i], sizeof(float) * xy);
    }
    hipMalloc(&f, sizeof(float*) * q);
    hipMalloc(&f_buffer, sizeof(float*) * q);

    hipMemcpy(f, f_host, sizeof(float) * q, hipMemcpyHostToDevice);
    hipMemcpy(f_buffer, f_host_buffer, sizeof(float) * q, hipMemcpyHostToDevice);
    free(f_host);
    free(f_host_buffer);
}


DatasetFP32::DatasetFP32(long long X, long long Y, Log& log, int d, int q) {
    mem_size = 0;
    bool success = true;
    hipError_t cuda_alloc;
    log.new_state("DatasetFP32");
    xy = X * Y;
    mem_size = xy *  (q * 2 + d + 3) * sizeof(float);
    log.log_state(std::to_string(mem_size));
    size_t dev_mem_avail, dev_mem_tot;
    hipMemGetInfo(&dev_mem_avail, &dev_mem_tot);
    log.log_state(std::to_string(dev_mem_avail));
    if(dev_mem_avail  < mem_size){
        log.new_state("Unable to allocate: not enough memory on the device");
        return;
    }
    if(dev_mem_tot  < mem_size){
        log.new_state("Unable to allocate: Exceeds total device memory");
        return;
    }
    // dist fnc alloc
    float** f_host = (float**)malloc( sizeof(float*) * q);
    float** f_host_buffer = (float**)malloc( sizeof(float*) * q);
    if((nullptr == f_host) || (f_host_buffer == nullptr)){
        free(f_host);
        free(f_host_buffer);
        goto exit;
    }
    log.log_mem(sizeof(float*) * q * 2);
    for(int i = 0 ; i < q; i ++){
        cuda_alloc = hipMalloc(&f_host[i], sizeof(float) * xy);
        cuda_alloc != hipSuccess ? success = false: success &= 1;
        cuda_alloc = hipMalloc(&f_host_buffer[i], sizeof(float) * xy);
        cuda_alloc != hipSuccess ? success = false: success &= 1;
        log.log_mem_dev(sizeof(float) * xy * 2);
    }
    cuda_alloc = hipMalloc(&f, sizeof(float*) * q);
    cuda_alloc != hipSuccess ? success = false: success &= 1;
    cuda_alloc = hipMalloc(&f_buffer, sizeof(float*) * q);
    cuda_alloc != hipSuccess ? success = false: success &= 1;
    log.log_mem_dev(sizeof(float*) * q * 2);

    cuda_alloc = hipMemcpy(f, f_host, sizeof(float*) * q, hipMemcpyHostToDevice);
    cuda_alloc != hipSuccess ? success = false: success &= 1;
    cuda_alloc = hipMemcpy(f_buffer, f_host_buffer, sizeof(float*) * q, hipMemcpyHostToDevice);
    cuda_alloc != hipSuccess ? success = false: success &= 1;

    //macroscopic alloc
    cuda_alloc = hipMalloc(&rho, sizeof(float) * xy);
    cuda_alloc != hipSuccess ? success = false: success &= 1;
    log.log_mem_dev(sizeof(float) * xy);
    cuda_alloc = hipMalloc(&p, sizeof(float) * xy);
    cuda_alloc != hipSuccess ? success = false: success &= 1;
    log.log_mem_dev(sizeof(float) * xy);
    cuda_alloc = hipMalloc(&v, sizeof(vec2FP32) * xy);
    cuda_alloc != hipSuccess ? success = false: success &= 1;
    log.log_mem_dev(sizeof(float) * xy);

    if(!success){
        log.log_state("initialization failed.");
        goto dealloc;
    }
    else{
        log.log_state("succeeded.");
        size_t  alloc_mem;
        hipMemGetInfo(&alloc_mem, &dev_mem_tot);
        log.log_state("Allocated " + std::to_string((-alloc_mem+dev_mem_avail) * 1E-6) + " mb on Device");
        free(f_host_buffer);
        free(f_host);
    }
    exit:
    return;

    dealloc:
    hipFree(rho);
    hipFree(p);
    hipFree(v);
    for(int i = 0; i < q; i ++){
        hipFree(f_host[i]);
        hipFree(f_host_buffer[i]);
    }
    hipFree(f);
    hipFree(f_buffer);
    free(f_host_buffer);
    free(f_host);
}


