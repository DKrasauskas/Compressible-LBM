#include "hip/hip_runtime.h"
//
// Created by domin on 09/05/2025.
//

#include "../Physics/Device.cuh"
#include "../Physics/Device.cuh"

void Device::info(Log &log) {
    log.new_state( "CUDA");
    log.log_state(std::string ("Device ") + this->prop.name);
    log.log_state(std::string ("Memory ") + std::to_string(this->mem_tot - this->mem_avail) + "MB used");
    log.log_state(std::string ("Memory ") + std::to_string(this->mem_avail) + "MB avail");
}
Device::Device(unsigned int id) {
    size_t net, avail;
    hipGetDeviceProperties(&this->prop, id);
    hipMemGetInfo(&avail, &net);
    this->mem_tot = (float)net * (1E-6);
    this->mem_avail = (float)avail * (1E-6);
}