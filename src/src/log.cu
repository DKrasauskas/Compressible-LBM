#include "hip/hip_runtime.h"
//
// Created by domin on 07/05/2025.
//
#include "../settings/log.h"


void Log::log_state(std::string data) {
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
    this->log_file << data << "\n";
}

void Log::begin() {
    this->log_file.open(this->output_path);
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
}
void Log::end() {
    this->log_file.close();
}

void Log::log_mem(unsigned int size) {
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
    this->log_file << "allocating " << size << "bytes" << "\n";
}
void Log::log_mem_dev(unsigned int size) {
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
    this->log_file << "allocating (device) " << size << "bytes" << "\n";
}
void Log::log_cuda_mem(std::string device, unsigned int total_mem, unsigned int free_mem) {
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
    this->log_file << "CUDA Properties\n" << "Device " << device << "\n VRAM " << total_mem << " total\n VRAM " << free_mem << " avail\n";
}
void Log::new_state() {
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
    this->log_file << "_____________________________________________________________" << "\n";
}

void Log::new_state(std::string name) {
    if (!log_file.is_open()) {
        throw std::runtime_error("Unable to open log file");
    }
    this->log_file<<
                   + "\n___________________________"
                   + name
                   + "______________________________\n"<< "\n";
}