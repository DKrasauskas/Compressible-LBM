#include "hip/hip_runtime.h"
#define uint unsigned int

#include <glad.h>
#include <glfw3.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cuda_gl_interop.h>

#include "src/settings/log.h"
Log program_log;


#include "src/settings/setup.cuh"
#include "src/Graphics/opengl/grid.h"
#include "src/Graphics/opengl/Shader.h"
#include "src/Graphics/opengl/buffer.h"


#include "include/imgui-master/imgui.h"
#include "include/imgui-master/backends/imgui_impl_opengl3.h"
#include "include/imgui-master/backends/imgui_impl_glfw.h"
#include "include/implot-master/implot.h"

#include "src/Physics/process.cuh"
#include "src/Graphics/opengl/controls.h"
#include "src/Graphics/opengl/opengl_context.h"

#include "src/Physics/GPU-code/physics.cuh"
#include "src/Physics/Device.cuh"
#include "src/Graphics/imgui/Plot.h"
#include "src/Physics/dataset.cuh"

__device__ Domain* D;


int main()
{
    program_log.output_path = "log.txt";
    program_log.begin();
    Device main_dev = Device(0);
    main_dev.info(program_log);
    GLFWwindow* window = init_OpenGL(program_log);
    GLFWwindow* plotting = init_OpenGL(1000, 500, "plot");
    glfwMakeContextCurrent(window);
    Shader vertex("../src/Graphics/opengl/GLSL/vertex.glsl", "../src/Graphics/opengl/GLSL/fragment.glsl", program_log);
    Shader compute("../src/Graphics/opengl/GLSL/compute.glsl", program_log);
    Grid gd = grid(2);
    Buffer buff((void*)gd.vertices, (void*)gd.indices, gd.v_size, gd.i_size);
    scene s = { &compute.ID, &vertex.ID, make_texture(), make_buffer() };
    Menu p(plotting);

    //LBM initialization
   // DatasetFP32(10000, 20000, program_log);
   // return 0;
   Domain* host = LBM::begin(&D);

    state = 2;
    int counter = 0;
    LBM::initial_conditions(&D, &host);
    while (!glfwWindowShouldClose(window) && !glfwWindowShouldClose(plotting))
    {
        if (counter > 1   && counter % 1  == 0){

            glfwMakeContextCurrent(window);
            renderScene(s, window, &buff, gd.v_size);
            gradient <<<blockH, threadH >>> (D, output, host->mask, NX, map_scale, state);
            hipDeviceSynchronize();
            // hipMemcpy(data, host->rho + 200 * 400, sizeof(float) * 400, hipMemcpyDeviceToHost);
            p.render_menu(plotting, host);
            if (p.rho && p.active[0]) {
               // upload_plot(host->rho + 200 * 400, p.plots[0].plotY, 400);

            }
            if(p.simulation_state)LBM::job(&D, &host, counter, counter);

        }
        else {
            std::cout << counter << "\n";
            //renderScene(s, window, &buff, gd.v_size);
            LBM::job(&D, &host, counter, counter);
        }
        counter++;

    }
    program_log.end();
    return 0;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}


